#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <functional>
#include <array>
#include <sys/time.h>

#include "kernels.cu"

//In open CL program ->
// #define MAX_THREADS_PER_BLOCK 256

#define MAX_THREADS_PER_BLOCK 512
int num_of_blocks;
int num_of_threads_per_block;

#define NUM_ITERATIONS 50

int edge_compare(const void *lhs, const void *rhs) {
	int l = ((Edge *)lhs)->in_vertex;
	int r = ((Edge *)rhs)->in_vertex;
	
	return (l - r);
}

int edge_compare_reverse(const void *lhs, const void *rhs) {
	int l = ((Edge *)lhs)->out_vertex;
	int r = ((Edge *)rhs)->out_vertex;
	
	return (l - r);
}

void run_pagerank_gpu_edgelist(int no_of_nodes, Node *h_graph_nodes, int edge_list_size, Edge *h_graph_edges, double* time_taken)
					throw(std::string) {

	float *h_pagerank = (float *) malloc (no_of_nodes*sizeof(float));
	float *h_pagerank_new = (float *) malloc (no_of_nodes*sizeof(float));
	for (int i=0; i< no_of_nodes; i++) {
		h_pagerank[i] = 0.25;
		h_pagerank_new[i] = 0.0;
	}
	
	//--1 transfer data from host to device

	float *d_pagerank;
	hipMalloc( (void**) &d_pagerank, sizeof(float)*no_of_nodes) ;
	hipMemcpy( d_pagerank, h_pagerank, sizeof(float)*no_of_nodes, hipMemcpyHostToDevice) ;

	float *d_pagerank_new;
	hipMalloc( (void**) &d_pagerank_new, sizeof(float)*no_of_nodes) ;
	hipMemcpy( d_pagerank_new, h_pagerank_new, sizeof(float)*no_of_nodes, hipMemcpyHostToDevice) ;

	Node *d_graph_nodes;
	hipMalloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes) ;
	hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice) ;

	Edge *d_graph_edges;
	hipMalloc( (void**) &d_graph_edges, sizeof(Edge)*edge_list_size) ;
	hipMemcpy( d_graph_edges, h_graph_edges, sizeof(Edge)*edge_list_size, hipMemcpyHostToDevice) ;

	int *d_no_of_edges;
	hipMalloc( (void**) &d_no_of_edges, sizeof(int)) ;
	hipMemcpy( d_no_of_edges, &edge_list_size, sizeof(int), hipMemcpyHostToDevice) ;

	int *d_no_of_nodes;
	hipMalloc( (void**) &d_no_of_nodes, sizeof(int)) ;
	hipMemcpy( d_no_of_nodes, &no_of_nodes, sizeof(int), hipMemcpyHostToDevice) ;

	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

	// int device;
	// hipGetDevice(&device);
	// struct hipDeviceProp_t properties;
	// hipGetDeviceProperties(&properties, device);
	// printf("using %d multiprocessors\n",properties.multiProcessorCount);
	// printf("max threads per processor: %d\n",properties.maxThreadsPerMultiProcessor);
	// printf("runing with dim3 num_of_blocks %d, num_of_threads_per_block %d\n", num_of_blocks, num_of_threads_per_block);

	try{
		int i = 0;
		int j=0;
		struct timeval t1, t2;
		double elapsedTime;
		// start timer
		gettimeofday(&t1, NULL);
		while(i < NUM_ITERATIONS){

			//Initialize the d_pagerank and d_pagerank_new arrays before invoking the kernel
			//Not required for the first iteration
			if (i!=0) {
				update_pagerank_arrays<<< grid, threads, 0 >>> (d_pagerank, d_pagerank_new, d_no_of_nodes);
			}
			hipError_t err = hipGetLastError();
			if ( hipSuccess != err )
			{
				fprintf( stderr, "cudaCheckError() for kernel launch of update_pagerank_array failed with error : %s\n",
							hipGetErrorString( err ) );
				exit( -1 );
			}
			
			hipDeviceSynchronize();

			edgelist<<< grid, threads, 0 >>>( 	d_graph_nodes,
												d_graph_edges, 
												d_no_of_edges,
												d_pagerank,
												d_pagerank_new);
			err = hipGetLastError();
			if ( hipSuccess != err )
			{
				fprintf( stderr, "cudaCheckError() for pagerank kernel launch failed with error : %s\n",
							hipGetErrorString( err ) );
				exit( -1 );
			}
			hipDeviceSynchronize(); 
			i++;
		}
		// stop timer
		gettimeofday(&t2, NULL);

		// compute and print the elapsed time in millisec
		elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000000000.0;      // sec to ns
		elapsedTime += (t2.tv_usec - t1.tv_usec) * 1000.0;   // us to ns
		printf("Kernel time : %f ns\n", elapsedTime);
		
		// hipError_t err = hipMemcpy((void *) h_pagerank_new, (void *) d_pagerank_new, no_of_nodes*sizeof(float), hipMemcpyDeviceToHost);
		// if (hipSuccess != err) {
		// 	fprintf(stderr, "memcopy new pagerank_new from device to host failed with error %s\n", hipGetErrorString(err));
		// 	exit(-1);
		// }
		// printf("New pageranks are : \n");
		// int max = 0;
		// for (int i=0; i<no_of_nodes; i++) {
		// 	printf("%d : %f, ", i, h_pagerank_new[i]);
		// }
		// printf("\n");
		
		//--4 release cuda resources.
		hipFree(d_graph_nodes);
		hipFree(d_graph_edges);
		hipFree(d_no_of_edges);
		hipFree(d_no_of_nodes);
		hipFree(d_pagerank);
		hipFree(d_pagerank_new);
	}
	catch(std::string msg){		
		hipFree(d_graph_nodes);
		hipFree(d_graph_edges);
		hipFree(d_no_of_edges);
		hipFree(d_no_of_nodes);
		hipFree(d_pagerank);
		hipFree(d_pagerank_new);
		std::string e_str = "in run_transpose_gpu -> ";
		e_str += msg;
		throw(e_str);
	}
	return ;
}

void run_pagerank_gpu_vertex_push(int no_of_nodes, Node* h_graph_nodes, int edge_list_size, Edge *h_graph_edges, int * h_neighbours, double *time_taken)
								throw(std::string) {

	float *h_pagerank = (float *) malloc (no_of_nodes*sizeof(float));
	float *h_pagerank_new = (float *) malloc (no_of_nodes*sizeof(float));
	for (int i=0; i< no_of_nodes; i++) {
		h_pagerank[i] = 0.25;
		h_pagerank_new[i] = 0.0;
	}
	
	//--1 transfer data from host to device

	float *d_pagerank;
	hipMalloc( (void**) &d_pagerank, sizeof(float)*no_of_nodes) ;
	hipMemcpy( d_pagerank, h_pagerank, sizeof(float)*no_of_nodes, hipMemcpyHostToDevice) ;

	float *d_pagerank_new;
	hipMalloc( (void**) &d_pagerank_new, sizeof(float)*no_of_nodes) ;
	hipMemcpy( d_pagerank_new, h_pagerank_new, sizeof(float)*no_of_nodes, hipMemcpyHostToDevice) ;

	Node *d_graph_nodes;
	hipMalloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes) ;
	hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice) ;

	Edge *d_graph_edges;
	hipMalloc( (void**) &d_graph_edges, sizeof(Edge)*edge_list_size) ;
	hipMemcpy( d_graph_edges, h_graph_edges, sizeof(Edge)*edge_list_size, hipMemcpyHostToDevice) ;

	int *d_neighbours;
	hipMalloc( (void**) &d_neighbours, sizeof(int)*edge_list_size) ;
	hipMemcpy( d_neighbours, h_neighbours, sizeof(int)*edge_list_size, hipMemcpyHostToDevice) ;

	int *d_no_of_edges;
	hipMalloc( (void**) &d_no_of_edges, sizeof(int)) ;
	hipMemcpy( d_no_of_edges, &edge_list_size, sizeof(int), hipMemcpyHostToDevice) ;

	int *d_no_of_nodes;
	hipMalloc( (void**) &d_no_of_nodes, sizeof(int)) ;
	hipMemcpy( d_no_of_nodes, &no_of_nodes, sizeof(int), hipMemcpyHostToDevice) ;

	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

	// int device;
	// hipGetDevice(&device);
	// struct hipDeviceProp_t properties;
	// hipGetDeviceProperties(&properties, device);
	// printf("using %d multiprocessors\n",properties.multiProcessorCount);
	// printf("max threads per processor: %d\n",properties.maxThreadsPerMultiProcessor);
	// printf("runing with dim3 num_of_blocks %d, num_of_threads_per_block %d\n", num_of_blocks, num_of_threads_per_block);

	try{
		int i = 0;
		struct timeval t1, t2;
		double elapsedTime;
		// start timer
		gettimeofday(&t1, NULL);
		while(i < NUM_ITERATIONS){

			//Initialize the d_pagerank and d_pagerank_new arrays before invoking the kernel
			//Not required for the first iteration
			if (i!=0) {
				update_pagerank_arrays<<< grid, threads, 0 >>> (d_pagerank, d_pagerank_new, d_no_of_nodes);
			}
			hipError_t err = hipGetLastError();
			if ( hipSuccess != err )
			{
				fprintf( stderr, "cudaCheckError() for kernel launch of update_pagerank_array failed with error : %s\n",
							hipGetErrorString( err ) );
				exit( -1 );
			}
			
			hipDeviceSynchronize();

			vertex_push<<< grid, threads, 0 >>>( 	d_graph_nodes,
												d_graph_edges, 
												d_no_of_edges,
												d_neighbours,
												d_pagerank,
												d_pagerank_new);
			err = hipGetLastError();
			if ( hipSuccess != err )
			{
				fprintf( stderr, "cudaCheckError() for pagerank kernel launch failed with error : %s\n",
							hipGetErrorString( err ) );
				exit( -1 );
			}
			hipDeviceSynchronize(); 
			i++;
		}
		// stop timer
		gettimeofday(&t2, NULL);

		// compute and print the elapsed time in millisec
		elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000000000.0;      // sec to ns
		elapsedTime += (t2.tv_usec - t1.tv_usec) * 1000.0;   // us to ns
		printf("Kernel time : %f ns\n", elapsedTime);
		
		// hipError_t err = hipMemcpy((void *) h_pagerank_new, (void *) d_pagerank_new, no_of_nodes*sizeof(float), hipMemcpyDeviceToHost);
		// if (hipSuccess != err) {
		// 	fprintf(stderr, "memcopy new pagerank_new from device to host failed with error %s\n", hipGetErrorString(err));
		// 	exit(-1);
		// }
		// printf("New pageranks are : \n");
		// int max = 0;
		// for (int i=0; i<no_of_nodes; i++) {
		// 	printf("%d : %f, ", i, h_pagerank_new[i]);
		// }
		// printf("\n");
		
		//--4 release cuda resources.
		hipFree(d_graph_nodes);
		hipFree(d_graph_edges);
		hipFree(d_no_of_edges);
		hipFree(d_no_of_nodes);
		hipFree(d_pagerank);
		hipFree(d_pagerank_new);
	}
	catch(std::string msg){		
		hipFree(d_graph_nodes);
		hipFree(d_graph_edges);
		hipFree(d_no_of_edges);
		hipFree(d_no_of_nodes);
		hipFree(d_pagerank);
		hipFree(d_pagerank_new);
		std::string e_str = "in run_transpose_gpu -> ";
		e_str += msg;
		throw(e_str);
	}
	return ;
}

void run_pagerank_gpu_vertex_pull(int no_of_nodes, Node* h_graph_nodes, int edge_list_size, Edge *h_graph_edges, int *h_reverse_neighbours, double *time_taken, char *h_graph_visited)
								throw(std::string) {

	float *h_pagerank = (float *) malloc (no_of_nodes*sizeof(float));
	float *h_pagerank_new = (float *) malloc (no_of_nodes*sizeof(float));
	for (int i=0; i< no_of_nodes; i++) {
		h_pagerank[i] = 0.25;
		h_pagerank_new[i] = 0.0;
	}
	
	//--1 transfer data from host to device

	float *d_pagerank;
	hipMalloc( (void**) &d_pagerank, sizeof(float)*no_of_nodes) ;
	hipMemcpy( d_pagerank, h_pagerank, sizeof(float)*no_of_nodes, hipMemcpyHostToDevice) ;

	float *d_pagerank_new;
	hipMalloc( (void**) &d_pagerank_new, sizeof(float)*no_of_nodes) ;
	hipMemcpy( d_pagerank_new, h_pagerank_new, sizeof(float)*no_of_nodes, hipMemcpyHostToDevice) ;

	Node *d_graph_nodes;
	hipMalloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes) ;
	hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice) ;

	Edge *d_graph_edges;
	hipMalloc( (void**) &d_graph_edges, sizeof(Edge)*edge_list_size) ;
	hipMemcpy( d_graph_edges, h_graph_edges, sizeof(Edge)*edge_list_size, hipMemcpyHostToDevice) ;

	int *d_reverse_neighbours;
	hipMalloc( (void**) &d_reverse_neighbours, sizeof(int)*edge_list_size) ;
	hipMemcpy( d_reverse_neighbours, h_reverse_neighbours, sizeof(int)*edge_list_size, hipMemcpyHostToDevice) ;

	int *d_no_of_edges;
	hipMalloc( (void**) &d_no_of_edges, sizeof(int)) ;
	hipMemcpy( d_no_of_edges, &edge_list_size, sizeof(int), hipMemcpyHostToDevice) ;

	int *d_no_of_nodes;
	hipMalloc( (void**) &d_no_of_nodes, sizeof(int)) ;
	hipMemcpy( d_no_of_nodes, &no_of_nodes, sizeof(int), hipMemcpyHostToDevice) ;

	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

	// int device;
	// hipGetDevice(&device);
	// struct hipDeviceProp_t properties;
	// hipGetDeviceProperties(&properties, device);
	// printf("using %d multiprocessors\n",properties.multiProcessorCount);
	// printf("max threads per processor: %d\n",properties.maxThreadsPerMultiProcessor);
	// printf("runing with dim3 num_of_blocks %d, num_of_threads_per_block %d\n", num_of_blocks, num_of_threads_per_block);

	try{
		int i = 0;
		struct timeval t1, t2;
		double elapsedTime;
		// start timer
		gettimeofday(&t1, NULL);
		while(i < NUM_ITERATIONS){

			//Initialize the d_pagerank and d_pagerank_new arrays before invoking the kernel
			//Not required for the first iteration
			if (i!=0) {
				update_pagerank_arrays<<< grid, threads, 0 >>> (d_pagerank, d_pagerank_new, d_no_of_nodes);
			}
			hipError_t err = hipGetLastError();
			if ( hipSuccess != err )
			{
				fprintf( stderr, "cudaCheckError() for kernel launch of update_pagerank_array failed with error : %s\n",
							hipGetErrorString( err ) );
				exit( -1 );
			}
			
			hipDeviceSynchronize();

			vertex_pull<<< grid, threads, 0 >>>( 	d_graph_nodes,
												d_graph_edges, 
												d_no_of_edges,
												d_reverse_neighbours,
												d_pagerank,
												d_pagerank_new);
			err = hipGetLastError();
			if ( hipSuccess != err )
			{
				fprintf( stderr, "cudaCheckError() for pagerank kernel launch failed with error : %s\n",
							hipGetErrorString( err ) );
				exit( -1 );
			}
			hipDeviceSynchronize(); 
			i++;
		}
		// stop timer
		gettimeofday(&t2, NULL);

		// compute and print the elapsed time in millisec
		elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000000000.0;      // sec to ns
		elapsedTime += (t2.tv_usec - t1.tv_usec) * 1000.0;   // us to ns
		printf("Kernel time : %f ns\n", elapsedTime);
		
		// hipError_t err = hipMemcpy((void *) h_pagerank_new, (void *) d_pagerank_new, no_of_nodes*sizeof(float), hipMemcpyDeviceToHost);
		// if (hipSuccess != err) {
		// 	fprintf(stderr, "memcopy new pagerank_new from device to host failed with error %s\n", hipGetErrorString(err));
		// 	exit(-1);
		// }
		// printf("New pageranks are : \n");
		// int max = 0;
		// for (int i=0; i<no_of_nodes; i++) {
		// 	printf("%d : %f, ", i, h_pagerank_new[i]);
		// }
		// printf("\n");
		
		//--4 release cuda resources.
		hipFree(d_graph_nodes);
		hipFree(d_graph_edges);
		hipFree(d_no_of_edges);
		hipFree(d_no_of_nodes);
		hipFree(d_pagerank);
		hipFree(d_pagerank_new);
	}
	catch(std::string msg){		
		hipFree(d_graph_nodes);
		hipFree(d_graph_edges);
		hipFree(d_no_of_edges);
		hipFree(d_no_of_nodes);
		hipFree(d_pagerank);
		hipFree(d_pagerank_new);
		std::string e_str = "in run_transpose_gpu -> ";
		e_str += msg;
		throw(e_str);
	}
	return ;
}

void Usage(int argc, char**argv){

fprintf(stderr,"Usage: %s <input_file>\n", argv[0]);

}

long read_and_return_no_of_nodes(char *filename) {
	long no_of_nodes = 0;
	std::ifstream fin;
	fin.open(filename);
	std::string line;
	int max = 0;
	int min = INT_MAX;
	// int min = 0;
	
	while (std::getline(fin, line)) {
		int node_index = std::stol(line);
		if (node_index > max) max = node_index;
		if (node_index < min) min = node_index;
	}

	no_of_nodes = max - min + 1;

	return no_of_nodes;
}

int read_and_return_no_of_edges(char *filename) 
{
	int no_of_edges = 0;
	std::ifstream fin;
	fin.open(filename);
	std::string line;
	
	while (std::getline(fin, line)) no_of_edges++;

	return no_of_edges; 
}

int main(int argc, char * argv[])
{
	long no_of_nodes;
	int edge_list_size;
	FILE *fp;
	Node* h_graph_nodes;
	char *h_graph_mask, *h_updating_graph_mask, *h_graph_visited;
	try{
		/* For now, read the input files directly instead of reading from i/o*/
		char *input_fe = "/var/scratch/alvarban/BSc_2k19/graphs/G500/graph500-10.e";
		char *input_fv = "/var/scratch/alvarban/BSc_2k19/graphs/G500/graph500-10.v";

		// char *input_fe = "trisha-file.e";
		// char *input_fv = "trisha-file.v";
	
		// char *input_fe = "/home/tanand/rodinia_3.1/graph500-10-superconnected.e";
		
		no_of_nodes = read_and_return_no_of_nodes(input_fv);
		// no_of_nodes = 1025;
		printf("Number of nodes read are : %d\n", no_of_nodes);
		edge_list_size = read_and_return_no_of_edges(input_fe);
		printf("Number of edges read are : %d\n", edge_list_size);
		
		//Read in Graph from a file
		fp = fopen(input_fe,"r");
		if(!fp){
		  printf("Error Reading EdgeGraph file\n");
		  return 0;
		}
		int source = 0;


		// allocate host memory
		h_graph_nodes = (Node*) malloc(sizeof(Node)*no_of_nodes);
		for (int i=0; i < no_of_nodes; i++) {
			h_graph_nodes[i].no_of_edges = 0;
			h_graph_nodes[i].starting = -1;
			h_graph_nodes[i].reverse_starting = -1;
			h_graph_nodes[i].no_of_reverse_edges = 0;
		}
		h_graph_mask = (char*) malloc(sizeof(char)*no_of_nodes);
		h_updating_graph_mask = (char*) malloc(sizeof(char)*no_of_nodes);
		h_graph_visited = (char*) malloc(sizeof(char)*no_of_nodes);
		for (int i=0; i<no_of_nodes; i++) {
			h_graph_visited[i] = false;
		}
	
		int start, edgeno;   
		
		Edge* h_graph_edges = (Edge*) malloc(sizeof(Edge)*edge_list_size);
		int neighbour_index = 0;
		for(int i=0; i < edge_list_size ; i++){
			int in_index, out_index;
			float cost; //for datagen
			fscanf(fp, "%d", &in_index);
			fscanf(fp, "%d", &out_index);
			// fscanf(fp, "%f", &cost); //only for datagen - delete for others
			h_graph_edges[i].in_vertex = in_index;
			h_graph_edges[i].out_vertex = out_index;
			//Update the number of neighbours of the node with index in_index;
			h_graph_nodes[in_index].no_of_edges++;
			// std::cout<<h_graph_edges[i].in_vertex<<" "<<h_graph_edges[i].out_vertex<<", read values are : "<<in_index<<" "<<out_index<<endl;
		}


		//Call edgelist and reverse edgelist here //

		
		//compute neighbours array for vertex push
		int qsort_size = sizeof(h_graph_edges) / sizeof(h_graph_edges[0]);
		qsort((void *) h_graph_edges, edge_list_size, sizeof(Edge), edge_compare);

		int* neighbours = (int *)malloc(edge_list_size * sizeof(int));


		int node_index = -1;
		for (int i=0; i < edge_list_size; i++) {
			// printf("%d, %d\n", h_graph_edges[i].in_vertex, h_graph_edges[i].out_vertex);
			if ((i==0) || (node_index != h_graph_edges[i].in_vertex)) {
				node_index = h_graph_edges[i].in_vertex;
				h_graph_nodes[node_index].starting = i;
				// printf("For %d, starting is %d in neighbours array\n", node_index, i);
			}
			neighbours[i] = h_graph_edges[i].out_vertex;
		}
		
		// printf("Neighbours array : \n");
		// for (int i=0; i < edge_list_size; i++) {
		// 	printf ("%d \n", neighbours[i]);
		// }

		// compute reverse neighbours (parents) for vertex pull
		qsort((void *) h_graph_edges, edge_list_size, sizeof(Edge), edge_compare_reverse);

		int* reverse_neighbours = (int *)malloc(edge_list_size * sizeof(int));

		node_index = -1;
		for (int i=0; i < edge_list_size; i++) {
			// printf("%d, %d\n", h_graph_edges[i].in_vertex, h_graph_edges[i].out_vertex);
			if ((i==0) || (node_index != h_graph_edges[i].out_vertex)) {
				// if(i!=0) std::cout<<node_index<<": starting-"<<h_graph_nodes[node_index].starting<<", reverse-starting-"<<h_graph_nodes[node_index].reverse_starting<<", num reverses-"<<h_graph_nodes[node_index].no_of_reverse_edges<<endl;
				node_index = h_graph_edges[i].out_vertex;
				h_graph_nodes[node_index].reverse_starting = i;
			}
			h_graph_nodes[node_index].no_of_reverse_edges++;
			reverse_neighbours[i] = h_graph_edges[i].in_vertex;
			// if (node_index == 0) std::cout<<reverse_neighbours[i]<<endl;
		}

		if(fp)
			fclose(fp);    
		double time_taken = 0;
		//---------------------------------------------------------
		//--gpu entry
		num_of_blocks = 1;
		// num_of_threads_per_block = no_of_nodes;
		num_of_threads_per_block = edge_list_size;
		//Make execution Parameters according to the number of edges
		//Distribute threads across multiple Blocks if necessary
		if(edge_list_size>MAX_THREADS_PER_BLOCK){
			num_of_blocks = (int)ceil(edge_list_size/(double)MAX_THREADS_PER_BLOCK); 
			num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
		}

		// int device;
		// hipGetDevice(&device);
		// struct hipDeviceProp_t properties;
		// hipGetDeviceProperties(&properties, device);
		// printf("using %d multiprocessors\n",properties.multiProcessorCount);
		// printf("max threads per processor: %d\n",properties.maxThreadsPerMultiProcessor);
		// printf("runing with dim3 num_of_blocks %d, num_of_threads_per_block %d\n", num_of_blocks, num_of_threads_per_block);
		
		std::cout<<"\nEdgelist Implementation"<<std::endl;
		for (int i=0; i<5; i++)
			run_pagerank_gpu_edgelist(no_of_nodes, h_graph_nodes,edge_list_size,h_graph_edges, &time_taken);	
			
		
		num_of_blocks = 1;
		// num_of_threads_per_block = no_of_nodes;
		num_of_threads_per_block = no_of_nodes;
		//Make execution Parameters according to the number of nodes
		//Distribute threads across multiple Blocks if necessary
		if(no_of_nodes>MAX_THREADS_PER_BLOCK){
			num_of_blocks = (int)ceil(no_of_nodes/(double)MAX_THREADS_PER_BLOCK); 
			num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
		}

		std::cout<<std::endl<<"Vertex Push Implementation"<<std::endl;
		for (int i=0; i<5; i++)
			run_pagerank_gpu_vertex_push(no_of_nodes,h_graph_nodes,edge_list_size,h_graph_edges, neighbours, &time_taken);
		std::cout<<std::endl<<"Vertex Pull Implementation"<<std::endl;
		for (int i=0; i<5; i++)	
			run_pagerank_gpu_vertex_pull(no_of_nodes,h_graph_nodes,edge_list_size,h_graph_edges, reverse_neighbours, &time_taken, h_graph_visited);	
		
		//release host memory		
		free(h_graph_nodes);
		free(h_graph_mask);
		free(h_updating_graph_mask);
		free(h_graph_visited);

	}
	catch(std::string msg){
		std::cout<<"--cambine: exception in main ->"<<msg<<std::endl;
		//release host memory
		free(h_graph_nodes);
		free(h_graph_mask);
		free(h_updating_graph_mask);
		free(h_graph_visited);		
	}
		
    return 0;
}
