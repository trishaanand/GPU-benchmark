#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <functional>
#include <array>
#include <sys/time.h>

#include "kernels.cu"

//In open CL program ->
// #define MAX_THREADS_PER_BLOCK 256

#define MAX_THREADS_PER_BLOCK 512
int num_of_blocks;
int num_of_threads_per_block;
int work_group_size;

//Structure to hold a node information
// typedef struct
// {
// 	int starting;
// 	int reverse_starting;
// 	int no_of_edges;
// 	int no_of_reverse_edges;
// }Node;

// typedef struct {
// 	int in_vertex;
// 	int out_vertex;
// }Edge;

bool edge_compare(Edge lhs, Edge rhs) {
	return (lhs.in_vertex < rhs.in_vertex);
}

bool edge_compare_reverse(Edge lhs, Edge rhs) {
	return (lhs.out_vertex < rhs.out_vertex);
}

//----------------------------------------------------------
//--breadth first search on GPUs - edgelist
//----------------------------------------------------------
void run_bfs_gpu_edgelist(int no_of_nodes, Node *h_graph_nodes, int edge_list_size, \
		Edge *h_graph_edges,  \
		char *h_graph_visited, double* time_taken)
					throw(std::string){

	//int number_elements = height*width;
	int h_depth = -1;
	char h_over;

	int *h_level = (int *) malloc (no_of_nodes*sizeof(int)); //store the current minimum depth seen by a node
	for (int i=0; i< no_of_nodes; i++) {
		h_level[i] = INT_MAX;	
		h_graph_visited[i] = false;
	}
	h_level[0] = 0;
	
	//--1 transfer data from host to device

	Node *d_graph_nodes;
	hipMalloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes) ;
	hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice) ;

	Edge *d_graph_edges;
	hipMalloc( (void**) &d_graph_edges, sizeof(Edge)*edge_list_size) ;
	hipMemcpy( d_graph_edges, h_graph_edges, sizeof(Edge)*edge_list_size, hipMemcpyHostToDevice) ;

	char *d_graph_visited;
	hipMalloc( (void**) &d_graph_visited, sizeof(char)*no_of_nodes) ;
	hipMemcpy( d_graph_visited, h_graph_visited, sizeof(char)*no_of_nodes, hipMemcpyHostToDevice) ;

	char *d_over;
	hipMalloc( (void**) &d_over, sizeof(char)) ;

	int *d_depth;
	hipMalloc( (void**) &d_depth, sizeof(int)) ;
	
	int *d_level;
	hipMalloc( (void**) &d_level, sizeof(int)*no_of_nodes) ;
	hipMemcpy( d_level, h_level, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice) ;

	int *d_no_of_edges;
	hipMalloc( (void**) &d_no_of_edges, sizeof(int)) ;
	hipMemcpy( d_no_of_edges, &edge_list_size, sizeof(int), hipMemcpyHostToDevice) ;

	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

	// int device;
	// hipGetDevice(&device);
	// struct hipDeviceProp_t properties;
	// hipGetDeviceProperties(&properties, device);
	// printf("using %d multiprocessors\n",properties.multiProcessorCount);
	// printf("max threads per processor: %d\n",properties.maxThreadsPerMultiProcessor);
	// printf("runing with dim3 num_of_blocks %d, num_of_threads_per_block %d\n", num_of_blocks, num_of_threads_per_block);

	try{
		h_depth = -1;
		struct timeval t1, t2;
		double elapsedTime;
		// start timer
		gettimeofday(&t1, NULL);
		do{
			h_over = false;
			h_depth = h_depth + 1;
			// printf("\nNew iterations : traversing current depth %d \n", h_depth);
			hipMemcpy( d_depth, &h_depth, sizeof(int), hipMemcpyHostToDevice) ;
			hipMemcpy( d_over, &h_over, sizeof(char), hipMemcpyHostToDevice) ;
			
			edgelist<<< grid, threads, 0 >>>( 	d_graph_nodes,
												d_graph_edges, 
												d_graph_visited, 
												d_no_of_edges,
												d_over,
												d_depth,
												d_level);
			hipError_t err = hipGetLastError();
			if ( hipSuccess != err )
			{
				fprintf( stderr, "cudaCheckError() for kernel launch failed with error : %s\n",
							hipGetErrorString( err ) );
				exit( -1 );
			}
			hipDeviceSynchronize(); 

			hipMemcpy( &h_over, d_over, sizeof(char), hipMemcpyDeviceToHost) ;
		}while(h_over);
		// stop timer
		gettimeofday(&t2, NULL);

		// compute and print the elapsed time in millisec
		elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000000000.0;      // sec to ns
		elapsedTime += (t2.tv_usec - t1.tv_usec) * 1000.0;   // us to ns
		printf("Kernel time : %f ns\n", elapsedTime);
		// printf("No of iterations : %d\n",h_depth);

		hipError_t err = hipMemcpy((void *) h_level, (void *) d_level, no_of_nodes*sizeof(int), hipMemcpyDeviceToHost);
		hipDeviceSynchronize(); 
		// printf("New depths are : \n");
		// int max = 0;
		// for (int i=0; i<no_of_nodes; i++) {
		// 	printf("%d : %d, ", i, h_level[i]);
		// 	if (h_level[i] != INT_MAX && h_level[i]>max) max = h_level[i];
		// }
		// printf("\nMaximum depth seen is %d\n",max);

		
		//--4 release cuda resources.
		hipFree(d_graph_nodes);
		hipFree(d_graph_edges);
		hipFree(d_graph_visited);
		hipFree(d_no_of_edges);
		hipFree(d_over);
		hipFree(d_depth);
		hipFree(d_level);
	}
	catch(std::string msg){		
		hipFree(d_graph_nodes);
		hipFree(d_graph_edges);
		hipFree(d_graph_visited);
		hipFree(d_no_of_edges);
		hipFree(d_over);
		hipFree(d_depth);
		hipFree(d_level);
		std::string e_str = "in run_transpose_gpu -> ";
		e_str += msg;
		throw(e_str);
	}
	return ;
}

//----------------------------------------------------------
//--breadth first search on GPUs - reverse edgelist
//----------------------------------------------------------
void run_bfs_gpu_reverse_edgelist(int no_of_nodes, Node *h_graph_nodes, int edge_list_size, \
	Edge *h_graph_edges,  \
	char *h_graph_visited, double* time_taken)
				throw(std::string){

//int number_elements = height*width;
int h_depth = -1;
char h_over;

int *h_level = (int *) malloc (no_of_nodes*sizeof(int)); //store the current minimum depth seen by a node
for (int i=0; i< no_of_nodes; i++) {
	h_level[i] = INT_MAX;	
	h_graph_visited[i] = false;
}
h_level[0] = 0;

//--1 transfer data from host to device

Node *d_graph_nodes;
hipMalloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes) ;
hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice) ;

Edge *d_graph_edges;
hipMalloc( (void**) &d_graph_edges, sizeof(Edge)*edge_list_size) ;
hipMemcpy( d_graph_edges, h_graph_edges, sizeof(Edge)*edge_list_size, hipMemcpyHostToDevice) ;

char *d_graph_visited;
hipMalloc( (void**) &d_graph_visited, sizeof(char)*no_of_nodes) ;
hipMemcpy( d_graph_visited, h_graph_visited, sizeof(char)*no_of_nodes, hipMemcpyHostToDevice) ;

char *d_over;
hipMalloc( (void**) &d_over, sizeof(char)) ;

int *d_depth;
hipMalloc( (void**) &d_depth, sizeof(int)) ;

int *d_level;
hipMalloc( (void**) &d_level, sizeof(int)*no_of_nodes) ;
hipMemcpy( d_level, h_level, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice) ;

int *d_no_of_edges;
hipMalloc( (void**) &d_no_of_edges, sizeof(int)) ;
hipMemcpy( d_no_of_edges, &edge_list_size, sizeof(int), hipMemcpyHostToDevice) ;


// setup execution parameters
// if(no_of_nodes>MAX_THREADS_PER_BLOCK){
// 	num_of_blocks = (int)ceil(no_of_nodes/(double)MAX_THREADS_PER_BLOCK); 
// 	num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
// }

dim3  grid( num_of_blocks, 1, 1);
dim3  threads( num_of_threads_per_block, 1, 1);

// int device;
// hipGetDevice(&device);
// struct hipDeviceProp_t properties;
// hipGetDeviceProperties(&properties, device);
// printf("using %d multiprocessors\n",properties.multiProcessorCount);
// printf("max threads per processor: %d\n",properties.maxThreadsPerMultiProcessor);
// printf("runing with dim3 num_of_blocks %d, num_of_threads_per_block %d\n", num_of_blocks, num_of_threads_per_block);

try{
	//First run BFS once with level 0 for all nodes who dont have reverse neighbours
	h_depth = -1;
	struct timeval t1, t2;
	double elapsedTime;
	// start timer
	gettimeofday(&t1, NULL);
	do{
		h_over = false;
		h_depth = h_depth + 1;
		// printf("\n\nNew iterations : traversing current depth %d \n", h_depth);
		hipMemcpy( d_depth, &h_depth, sizeof(int), hipMemcpyHostToDevice) ;
		hipMemcpy( d_over, &h_over, sizeof(char), hipMemcpyHostToDevice) ;
		
		reverse_edgelist<<< grid, threads, 0 >>>( 	d_graph_nodes,
											d_graph_edges, 
											d_graph_visited, 
											d_no_of_edges,
											d_over,
											d_depth,
											d_level);
		hipError_t err = hipGetLastError();
		if ( hipSuccess != err )
		{
			fprintf( stderr, "cudaCheckError() for kernel launch failed with error : %s\n",
						hipGetErrorString( err ) );
			exit( -1 );
		}
		hipDeviceSynchronize(); 

		hipMemcpy( &h_over, d_over, sizeof(char), hipMemcpyDeviceToHost) ;
	}while(h_over);
	// stop timer
	gettimeofday(&t2, NULL);

	// compute and print the elapsed time in millisec
	// compute and print the elapsed time in millisec
	elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000000000.0;      // sec to ns
	elapsedTime += (t2.tv_usec - t1.tv_usec) * 1000.0;   // us to ns
	printf("Kernel time : %f ns\n", elapsedTime);
	
	// printf("No of iterations : %d\n",h_depth);

	// hipMemcpy(h_level, d_level, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice);
	// hipDeviceSynchronize(); 
	// printf("New depths are : \n");
	// int max = 0;
	// for (int i=0; i<no_of_nodes; i++) {
	// 	printf("%d : %d, ", i, h_level[i]);
	// 	if (h_level[i] != INT_MAX && h_level[i]>max) max = h_level[i];
	// }
	// printf("\nMaximum depth seen is %d\n",max);

	//--4 release cuda resources.
	hipFree(d_graph_nodes);
	hipFree(d_graph_edges);
	hipFree(d_graph_visited);
	hipFree(d_no_of_edges);
	hipFree(d_over);
	hipFree(d_depth);
	hipFree(d_level);
}
catch(std::string msg){		
	hipFree(d_graph_nodes);
	hipFree(d_graph_edges);
	hipFree(d_graph_visited);
	hipFree(d_no_of_edges);
	hipFree(d_over);
	hipFree(d_depth);
	hipFree(d_level);
	std::string e_str = "in run_transpose_gpu -> ";
	e_str += msg;
	throw(e_str);
}
return ;
}

// //----------------------------------------------------------
// //--breadth first search on GPUs - vertex push
// //----------------------------------------------------------
// void run_bfs_gpu_vertex_push(int no_of_nodes, Node* h_graph_nodes, int edge_list_size, Edge *h_graph_edges, int * h_neighbours, double *time_taken, char *h_graph_visited)
// 								throw(std::string) {
// 	int h_depth = -1;
// 	char h_over;

// 	int *h_level = (int *) malloc (no_of_nodes*sizeof(int)); //store the current minimum depth seen by a node
	
// 	for (int i=0; i< no_of_nodes; i++) {
// 		if (h_graph_nodes[i].no_of_reverse_edges == 0) {
// 			//if no node connects to the current node, set the level to zero
// 			h_level[i] = 0;
// 			h_graph_visited[i] = true;
// 		} else {
// 			h_level[i] = INT_MAX;	
// 		}
// 	}
	
// 	cl_mem d_graph_nodes, d_graph_edges, d_over, d_depth, d_level, d_neighbours, d_graph_visited;
// 	try{
// 		//--1 transfer data from host to device
// 		_clInit();	
// 		d_graph_nodes = _clMalloc(no_of_nodes*sizeof(Node), h_graph_nodes);
// 		d_graph_edges = _clMalloc(edge_list_size*sizeof(Edge), h_graph_edges);
// 		d_neighbours = _clMalloc(edge_list_size*sizeof(int), h_neighbours);
// 		d_graph_visited = _clMallocRW(no_of_nodes*sizeof(char), h_graph_visited);
		
// 		d_over = _clMallocRW(sizeof(char), &h_over);
// 		d_depth = _clMallocRW(sizeof(int), &h_depth);

// 		d_level = _clMallocRW(no_of_nodes*sizeof(int), h_level);
		
// 		_clMemcpyH2D(d_graph_nodes, no_of_nodes*sizeof(Node), h_graph_nodes);
// 		_clMemcpyH2D(d_graph_edges, edge_list_size*sizeof(Edge), h_graph_edges);	
// 		_clMemcpyH2D(d_level, no_of_nodes*sizeof(int), h_level);
// 		_clMemcpyH2D(d_neighbours, edge_list_size*sizeof(int), h_neighbours);
// 		_clMemcpyH2D(d_graph_visited, no_of_nodes*sizeof(char), h_graph_visited);
			
// 		//--2 invoke kernel
// #ifdef	PROFILING
// 		timer kernel_timer;
// 		double kernel_time = 0.0;		
// 		kernel_timer.reset();
// 		kernel_timer.start();
// #endif
		
// 		//First run BFS once with level 0 for all nodes who dont have reverse neighbours
// 		h_depth = -1;
// 		do{
// 			h_over = false;
// 			h_depth = h_depth + 1;
			
// 			_clMemcpyH2D(d_over, sizeof(char), &h_over);
// 			_clMemcpyH2D(d_depth, sizeof(int), &h_depth);
// 			//--kernel 0
// 			int kernel_id = 4;
// 			int kernel_idx = 0;
// 			_clSetArgs(kernel_id, kernel_idx++, d_graph_nodes);
// 			_clSetArgs(kernel_id, kernel_idx++, d_graph_edges);
// 			_clSetArgs(kernel_id, kernel_idx++, &no_of_nodes, sizeof(int));
// 			_clSetArgs(kernel_id, kernel_idx++, d_over);
// 			_clSetArgs(kernel_id, kernel_idx++, d_depth);
// 			_clSetArgs(kernel_id, kernel_idx++, d_level);
// 			_clSetArgs(kernel_id, kernel_idx++, d_neighbours);
// 			_clSetArgs(kernel_id, kernel_idx++, d_graph_visited);
			
// 			//int work_items = no_of_nodes;
// 			_clInvokeKernel(kernel_id, no_of_nodes, work_group_size);
			
// 			_clMemcpyD2H(d_over,sizeof(char), &h_over);
// 		}while(h_over);
			
// 		//Now run BFS for all the graph nodes which were disconnected from the above.
// 		int num_of_starting_points = 0;
// 		for (int i=0; i<no_of_nodes; i++) {
// 			if (h_graph_visited[i] != true) {
// 				num_of_starting_points++;
// 				//Start BFS from the ith node
// 				h_level[i] = 0;
// 				h_graph_visited[i] = true;
// 				_clMemcpyH2D(d_graph_visited, no_of_nodes*sizeof(char), h_graph_visited);
// 				_clMemcpyH2D(d_level, no_of_nodes*sizeof(int), h_level);
// 				h_depth = -1;
				
// 				do{
// 					h_over = false;
// 					h_depth = h_depth + 1;
					
// 					_clMemcpyH2D(d_over, sizeof(char), &h_over);
// 					_clMemcpyH2D(d_depth, sizeof(int), &h_depth);
// 					//--kernel 0
// 					int kernel_id = 4;
// 					int kernel_idx = 0;
// 					_clSetArgs(kernel_id, kernel_idx++, d_graph_nodes);
// 					_clSetArgs(kernel_id, kernel_idx++, d_graph_edges);
// 					_clSetArgs(kernel_id, kernel_idx++, &no_of_nodes, sizeof(int));
// 					_clSetArgs(kernel_id, kernel_idx++, d_over);
// 					_clSetArgs(kernel_id, kernel_idx++, d_depth);
// 					_clSetArgs(kernel_id, kernel_idx++, d_level);
// 					_clSetArgs(kernel_id, kernel_idx++, d_neighbours);
// 					_clSetArgs(kernel_id, kernel_idx++, d_graph_visited);
					
// 					//int work_items = no_of_nodes;
// 					_clInvokeKernel(kernel_id, no_of_nodes, work_group_size);
					
// 					_clMemcpyD2H(d_over,sizeof(char), &h_over);
// 				}while(h_over);

// 				//Update the h_graph_visited array
// 				_clMemcpyD2H(d_graph_visited,no_of_nodes*sizeof(char), h_graph_visited);
// 				//Update the h_level array
// 				_clMemcpyD2H(d_level,no_of_nodes*sizeof(int), h_level);
// 			}
// 		}

// 		_clFinish();
// 		// std::cout<<"Num iterations : "<<h_depth<<std::endl;
// #ifdef	PROFILING
// 		kernel_timer.stop();
// 		kernel_time = kernel_timer.getTimeInSeconds();
// 		*time_taken = kernel_time;
			
// #endif
// 		//--3 transfer data from device to host
// 		_clMemcpyD2H(d_level,no_of_nodes*sizeof(int), h_level);
// 		std::cout<<"New depths are : "<<std::endl;
// 		int max = 0;
// 		for (int i=0; i<no_of_nodes; i++) {
// 			std::cout<<i<<" : "<<h_level[i]<<", ";
// 			if (h_level[i]>max) max = h_level[i];
// 		}
// 		std::cout<<std::endl;
// 		std::cout<<"Maximum depth seen is "<<max<<std::endl;

// 		std::cout<<"kernel time(s):"<<kernel_time<<std::endl;	
// 		//--4 release cl resources.
// 		_clFree(d_graph_nodes);
// 		_clFree(d_graph_edges);
// 		_clFree(d_over);
// 		_clRelease();
// 	}
// 	catch(std::string msg){		
// 		_clFree(d_graph_nodes);
// 		_clFree(d_graph_edges);
// 		_clFree(d_over);
// 		_clRelease();
// 		std::string e_str = "in run_transpose_gpu -> ";
// 		e_str += msg;
// 		throw(e_str);
// 	}
// 	return ;
// }

// //----------------------------------------------------------
// //--breadth first search on GPUs - vertex pull
// //----------------------------------------------------------
// void run_bfs_gpu_vertex_pull(int no_of_nodes, Node* h_graph_nodes, int edge_list_size, Edge *h_graph_edges, int *h_reverse_neighbours, double *time_taken, char *h_graph_visited)
// 								throw(std::string) {
// 	int h_depth = -1;
// 	char h_over;

// 	int *h_level = (int *) malloc (no_of_nodes*sizeof(int)); //store the current minimum depth seen by a node
// 	for (int i=0; i< no_of_nodes; i++) {
// 		if (h_graph_nodes[i].no_of_reverse_edges == 0) {
// 			//if no node connects to the current node, set the level to zero
// 			h_level[i] = 0;
// 			h_graph_visited[i] = true;
// 		} else {
// 			h_level[i] = INT_MAX;	
// 		}
// 	}
	
// 	cl_mem d_graph_nodes, d_graph_edges, d_over, d_depth, d_level, d_reverse_neighbours, d_graph_visited;
// 	try{
// 		//--1 transfer data from host to device
// 		_clInit();	
// 		d_graph_nodes = _clMalloc(no_of_nodes*sizeof(Node), h_graph_nodes);
// 		d_graph_edges = _clMalloc(edge_list_size*sizeof(Edge), h_graph_edges);
// 		d_reverse_neighbours = _clMalloc(edge_list_size*sizeof(int), h_reverse_neighbours);
// 		d_graph_visited = _clMallocRW(no_of_nodes*sizeof(char), h_graph_visited);
		
// 		d_over = _clMallocRW(sizeof(char), &h_over);
// 		d_depth = _clMallocRW(sizeof(int), &h_depth);

// 		d_level = _clMallocRW(no_of_nodes*sizeof(int), h_level);
		
// 		_clMemcpyH2D(d_graph_nodes, no_of_nodes*sizeof(Node), h_graph_nodes);
// 		_clMemcpyH2D(d_graph_edges, edge_list_size*sizeof(Edge), h_graph_edges);	
// 		_clMemcpyH2D(d_level, no_of_nodes*sizeof(int), h_level);
// 		_clMemcpyH2D(d_reverse_neighbours, edge_list_size*sizeof(int), h_reverse_neighbours);
// 		_clMemcpyH2D(d_graph_visited, no_of_nodes*sizeof(char), h_graph_visited);
			
// 		//--2 invoke kernel
// #ifdef	PROFILING
// 		timer kernel_timer;
// 		double kernel_time = 0.0;		
// 		kernel_timer.reset();
// 		kernel_timer.start();
// #endif
// 		//First run BFS once with level 0 for all nodes who dont have reverse neighbours
// 		h_depth = -1;
		
// 		do{
// 			h_over = false;
// 			h_depth = h_depth + 1;
// 			if (!h_over)
// 				// std::cout<<"Start of iteration, h_over is "<<h_over<<", and h_depth is "<<h_depth<<std::endl;
// 			_clMemcpyH2D(d_over, sizeof(char), &h_over);
// 			_clMemcpyH2D(d_depth, sizeof(int), &h_depth);
// 			//--kernel 0
// 			int kernel_id = 5;
// 			int kernel_idx = 0;
// 			_clSetArgs(kernel_id, kernel_idx++, d_graph_nodes);
// 			_clSetArgs(kernel_id, kernel_idx++, d_graph_edges);
// 			_clSetArgs(kernel_id, kernel_idx++, &no_of_nodes, sizeof(int));
// 			_clSetArgs(kernel_id, kernel_idx++, d_over);
// 			_clSetArgs(kernel_id, kernel_idx++, d_depth);
// 			_clSetArgs(kernel_id, kernel_idx++, d_level);
// 			_clSetArgs(kernel_id, kernel_idx++, d_reverse_neighbours);
// 			_clSetArgs(kernel_id, kernel_idx++, d_graph_visited);
			
// 			//int work_items = no_of_nodes;
// 			_clInvokeKernel(kernel_id, no_of_nodes, work_group_size);
			
// 			_clMemcpyD2H(d_over,sizeof(char), &h_over);
// 			// if (h_over)
// 			// 	std::cout<<"End of iteration, h_over is "<<h_over<<std::endl;
// 			// else
// 			// 	std::cout<<"End of do-while loop"<<std::endl;
// 		}while(h_over);

// 		//Update the h_graph_visited array
// 		_clMemcpyD2H(d_graph_visited,no_of_nodes*sizeof(char), h_graph_visited);
// 		//Update the h_level array
// 		_clMemcpyD2H(d_level,no_of_nodes*sizeof(int), h_level);

// 		//Now run BFS for all the graph nodes which were disconnected from the above.
// 		int num_of_starting_points = 0;
// 		for (int i=0; i<no_of_nodes; i++) {
// 			if (h_graph_visited[i] != true) {
// 				num_of_starting_points++;
// 				//Start BFS from the ith node
// 				// std::cout<<"Going to run BFS again because node "<<i<<" has not been visited yet"<<std::endl;
// 				h_level[i] = 0;
// 				h_graph_visited[i] = true;
// 				_clMemcpyH2D(d_graph_visited, no_of_nodes*sizeof(char), h_graph_visited);
// 				_clMemcpyH2D(d_level, no_of_nodes*sizeof(int), h_level);
// 				h_depth = -1;
				
// 				do{
// 					h_over = false;
// 					h_depth = h_depth + 1;
// 					// if (!h_over)
// 					// 	std::cout<<"Start of iteration, h_over is "<<h_over<<", and h_depth is "<<h_depth<<std::endl;
// 					_clMemcpyH2D(d_over, sizeof(char), &h_over);
// 					_clMemcpyH2D(d_depth, sizeof(int), &h_depth);
// 					//--kernel 0
// 					int kernel_id = 5;
// 					int kernel_idx = 0;
// 					_clSetArgs(kernel_id, kernel_idx++, d_graph_nodes);
// 					_clSetArgs(kernel_id, kernel_idx++, d_graph_edges);
// 					_clSetArgs(kernel_id, kernel_idx++, &no_of_nodes, sizeof(int));
// 					_clSetArgs(kernel_id, kernel_idx++, d_over);
// 					_clSetArgs(kernel_id, kernel_idx++, d_depth);
// 					_clSetArgs(kernel_id, kernel_idx++, d_level);
// 					_clSetArgs(kernel_id, kernel_idx++, d_reverse_neighbours);
// 					_clSetArgs(kernel_id, kernel_idx++, d_graph_visited);
					
// 					//int work_items = no_of_nodes;
// 					_clInvokeKernel(kernel_id, no_of_nodes, work_group_size);
					
// 					_clMemcpyD2H(d_over,sizeof(char), &h_over);
// 					// if (h_over)
// 					// 	std::cout<<"End of iteration, h_over is "<<h_over<<std::endl;
// 					// else
// 					// 	std::cout<<"End of do-while loop"<<std::endl;
// 				}while(h_over);

// 				//Update the h_graph_visited array
// 				_clMemcpyD2H(d_graph_visited,no_of_nodes*sizeof(char), h_graph_visited);
// 				//Update the h_level array
// 				_clMemcpyD2H(d_level,no_of_nodes*sizeof(int), h_level);
// 			}
// 		}
// 		std::cout<<"Number of starting points : "<<num_of_starting_points<<std::endl;
// 		_clFinish();
		
// #ifdef	PROFILING
// 		kernel_timer.stop();
// 		kernel_time = kernel_timer.getTimeInSeconds();
// 		*time_taken = kernel_time;
			
// #endif
// 		_clMemcpyD2H(d_level,no_of_nodes*sizeof(int), h_level);
// 		std::cout<<"New depths are : "<<std::endl;
// 		int max = 0;
// 		for (int i=0; i<no_of_nodes; i++) {
// 			if (h_level[i] != INT_MAX) std::cout<<i<<" : "<<h_level[i]<<", ";
// 			if (h_level[i]>max) max = h_level[i];
// 		}
// 		std::cout<<std::endl;
// 		std::cout<<"Maximum depth seen is "<<max<<std::endl;


// 		std::cout<<"kernel time(s):"<<kernel_time<<std::endl;	
// 		//--4 release cl resources.
// 		_clFree(d_graph_nodes);
// 		_clFree(d_graph_edges);
// 		_clFree(d_over);
// 		_clRelease();
// 	}
// 	catch(std::string msg){		
// 		_clFree(d_graph_nodes);
// 		_clFree(d_graph_edges);
// 		_clFree(d_over);
// 		_clRelease();
// 		std::string e_str = "in run_transpose_gpu -> ";
// 		e_str += msg;
// 		throw(e_str);
// 	}
// 	return ;
// }

void Usage(int argc, char**argv){

fprintf(stderr,"Usage: %s <input_file>\n", argv[0]);

}

long read_and_return_no_of_nodes(char *filename) {
	long no_of_nodes = 0;
	std::ifstream fin;
	fin.open(filename);
	std::string line;
	int max = 0;
	int min = INT_MAX;
	// int min = 0;
	
	while (std::getline(fin, line)) {
		int node_index = std::stol(line);
		if (node_index > max) max = node_index;
		if (node_index < min) min = node_index;
	}

	no_of_nodes = max - min + 1;

	return no_of_nodes;
}

int read_and_return_no_of_edges(char *filename) 
{
	int no_of_edges = 0;
	std::ifstream fin;
	fin.open(filename);
	std::string line;
	
	while (std::getline(fin, line)) no_of_edges++;

	return no_of_edges; 
}
//----------------------------------------------------------
//--cambine:	main function
//--author:		created by Jianbin Fang
//--date:		25/01/2011
//----------------------------------------------------------
int main(int argc, char * argv[])
{
	long no_of_nodes;
	int edge_list_size;
	FILE *fp;
	Node* h_graph_nodes;
	char *h_graph_mask, *h_updating_graph_mask, *h_graph_visited;
	try{
		/* For now, read the input files directly instead of reading from i/o*/
		char *input_fe = "/var/scratch/alvarban/BSc_2k19/graphs/G500/graph500-10.e.edges";
		char *input_fv = "/var/scratch/alvarban/BSc_2k19/graphs/G500/graph500-10.v";

		// char *input_fe = "trisha-file.e";
		// char *input_fv = "trisha-file.v";
	
		// char *input_fe = "/home/tanand/rodinia_3.1/graph500-10-superconnected.e";
		
		no_of_nodes = read_and_return_no_of_nodes(input_fv);
		// no_of_nodes = 1025;
		printf("Number of nodes read are : %d\n", no_of_nodes);
		edge_list_size = read_and_return_no_of_edges(input_fe);
		printf("Number of edges read are : %d\n", edge_list_size);
		
		//Read in Graph from a file
		fp = fopen(input_fe,"r");
		if(!fp){
		  printf("Error Reading EdgeGraph file\n");
		  return 0;
		}
		int source = 0;

		num_of_blocks = 1;
		// num_of_threads_per_block = no_of_nodes;
		num_of_threads_per_block = edge_list_size;

		//Make execution Parameters according to the number of nodes
		//Distribute threads across multiple Blocks if necessary
		if(edge_list_size>MAX_THREADS_PER_BLOCK){
			num_of_blocks = (int)ceil(edge_list_size/(double)MAX_THREADS_PER_BLOCK); 
			num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
		}
		work_group_size = num_of_threads_per_block;
		// allocate host memory
		h_graph_nodes = (Node*) malloc(sizeof(Node)*no_of_nodes);
		for (int i=0; i < no_of_nodes; i++) {
			h_graph_nodes[i].no_of_edges = 0;
			h_graph_nodes[i].starting = -1;
			h_graph_nodes[i].reverse_starting = -1;
			h_graph_nodes[i].no_of_reverse_edges = 0;
		}
		h_graph_mask = (char*) malloc(sizeof(char)*no_of_nodes);
		h_updating_graph_mask = (char*) malloc(sizeof(char)*no_of_nodes);
		h_graph_visited = (char*) malloc(sizeof(char)*no_of_nodes);
		for (int i=0; i<no_of_nodes; i++) {
			h_graph_visited[i] = false;
		}
	
		int start, edgeno;   
		
		Edge* h_graph_edges = (Edge*) malloc(sizeof(Edge)*edge_list_size);
		int neighbour_index = 0;
		for(int i=0; i < edge_list_size ; i++){
			int in_index, out_index;
			float cost; //for datagen
			fscanf(fp, "%d", &in_index);
			fscanf(fp, "%d", &out_index);
			// fscanf(fp, "%f", &cost); //only for datagen - delete for others
			h_graph_edges[i].in_vertex = in_index;
			h_graph_edges[i].out_vertex = out_index;
			//Update the number of neighbours of the node with index in_index;
			h_graph_nodes[in_index].no_of_edges++;
			// std::cout<<h_graph_edges[i].in_vertex<<" "<<h_graph_edges[i].out_vertex<<", read values are : "<<in_index<<" "<<out_index<<endl;
		}

		//compute neighbours array for vertex push
		// std::sort(h_graph_edges, h_graph_edges+edge_list_size, edge_compare);

		// int* neighbours = (int *)malloc(edge_list_size * sizeof(int));

		// int node_index = -1;
		// for (int i=0; i < edge_list_size; i++) {
		// 	// printf("%d, %d\n", h_graph_edges[i].in_vertex, h_graph_edges[i].out_vertex);
		// 	if ((i==0) || (node_index != h_graph_edges[i].in_vertex)) {
		// 		node_index = h_graph_edges[i].in_vertex;
		// 		h_graph_nodes[node_index].starting = i;
		// 		// printf("For %d, starting is %d in neighbours array\n", node_index, i);
		// 	}
		// 	neighbours[i] = h_graph_edges[i].out_vertex;
		// }
		
		// // printf("Neighbours array : \n");
		// // for (int i=0; i < edge_list_size; i++) {
		// // 	printf ("%d \n", neighbours[i]);
		// // }

		// //compute reverse neighbours (parents) for vertex pull
		// std::sort(h_graph_edges, h_graph_edges+edge_list_size, edge_compare_reverse);

		// int* reverse_neighbours = (int *)malloc(edge_list_size * sizeof(int));

		// node_index = -1;
		// for (int i=0; i < edge_list_size; i++) {
		// 	// std::cout<<h_graph_edges[i].out_vertex<<", "<<h_graph_edges[i].in_vertex<<endl;
		// 	if ((i==0) || (node_index != h_graph_edges[i].out_vertex)) {
		// 		// if(i!=0) std::cout<<node_index<<": starting-"<<h_graph_nodes[node_index].starting<<", reverse-starting-"<<h_graph_nodes[node_index].reverse_starting<<", num reverses-"<<h_graph_nodes[node_index].no_of_reverse_edges<<endl;
		// 		node_index = h_graph_edges[i].out_vertex;
		// 		h_graph_nodes[node_index].reverse_starting = i;
		// 	}
		// 	h_graph_nodes[node_index].no_of_reverse_edges++;
		// 	reverse_neighbours[i] = h_graph_edges[i].in_vertex;
		// 	// if (node_index == 0) std::cout<<reverse_neighbours[i]<<endl;
		// }

		if(fp)
			fclose(fp);    
		double time_taken = 0;
		//---------------------------------------------------------
		//--gpu entry
		std::cout<<"Edgelist Implementation"<<std::endl;
		for (int i=0; i<5; i++)
			run_bfs_gpu_edgelist(no_of_nodes, h_graph_nodes,edge_list_size,h_graph_edges, h_graph_visited, &time_taken);	
		std::cout<<std::endl<<"Reverse Edgelist Implementation"<<std::endl;
		for (int i=0; i<5; i++)
			run_bfs_gpu_reverse_edgelist(no_of_nodes,h_graph_nodes,edge_list_size,h_graph_edges, h_graph_visited, &time_taken);	
		// std::cout<<"Vertex Push Implementation"<<std::endl;
		// for (int i=0; i<5; i++)
			// run_bfs_gpu_vertex_push(no_of_nodes,h_graph_nodes,edge_list_size,h_graph_edges, neighbours, &time_taken, h_graph_visited);
		// std::cout<<"Vertex Pull Implementation"<<std::endl;
		// for (int i=0; i<5; i++)	
			// run_bfs_gpu_vertex_pull(no_of_nodes,h_graph_nodes,edge_list_size,h_graph_edges, reverse_neighbours, &time_taken, h_graph_visited);	
		
		//release host memory		
		free(h_graph_nodes);
		free(h_graph_mask);
		free(h_updating_graph_mask);
		free(h_graph_visited);

	}
	catch(std::string msg){
		std::cout<<"--cambine: exception in main ->"<<msg<<std::endl;
		//release host memory
		free(h_graph_nodes);
		free(h_graph_mask);
		free(h_updating_graph_mask);
		free(h_graph_visited);		
	}
		
    return 0;
}
